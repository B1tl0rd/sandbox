#include "hip/hip_runtime.h"
/* based on NVIDIAs simpleGL example */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <cutil_inline.h>
#include <cutil_gl_inline.h>
#include <cutil_gl_error.h>
#include <hip/hip_vector_types.h>

#include <gsl/gsl_rng.h>

/* OpenGL */
static const int window_width = 800, window_height = 600;
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0., rotate_y = 0.;
float translate_z = -3.;

/* vertex buffer object */
GLuint vbo_pos;
hipGraphicsResource_t vbo_res;

/* check for CUDA error */
#define CHECK_ERROR check_cuda_error(__LINE__-1, __FILE__)

/* #bodies */
static int N;

/* #threads/block (leapfrog) */
static int TPB = 16;

/* #tiles (acceleration kernel) */
static int P;

/* softening factor (square), G, \Delta t */
static float EPS = 1., G = 10., DELTA_T = 0.01;
static float DAMPENING = .7;

/* acceleration */
__device__ float4 *a;
/* x,y,z: position; w: mass */
static float4 *r_host;
__device__ float4 *r;
/* velocity */
static float4 *v_host;
__device__ float4 *v;

/* GSL rng */
const gsl_rng_type *T;
gsl_rng *rng;

/* check for CUDA error */
static void check_cuda_error(const int line, const char *file)
{
	hipError_t e;

	e = hipGetLastError();
	if (e != hipSuccess) {
		printf("CUDA error: %s, line %i, file '%s'\n",
		       hipGetErrorString(e), line, file);
		exit(1);
	}
}

/* leap frog integration kernel (1 particle/thread) */
__global__ void leap_frog_1p_2(float4 *a, float4 *v, float4 *r, float delta_t,
                               float dampening)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	float3 v_tmp;

	v_tmp.x = v[i].x;
	v_tmp.y = v[i].y;
	v_tmp.z = v[i].z;

	v_tmp.x += a[i].x * delta_t;
	v_tmp.y += a[i].y * delta_t;
	v_tmp.z += a[i].z * delta_t;

	v_tmp.x *= dampening;
	v_tmp.y *= dampening;
	v_tmp.z *= dampening;

	r[i].x += v_tmp.x * delta_t;
	r[i].y += v_tmp.y * delta_t;
	r[i].z += v_tmp.z * delta_t;

	v[i] = make_float4(v_tmp.x, v_tmp.y, v_tmp.z, 0.f);
}

/* body-body interaction, returns a_i */
__device__ float3 interaction(float3 ri, float4 rj, float eps)
{
	float3 rij, ai;
	float dst_sqr, cube, inv_sqrt;

	/* distance vector */
	rij.x = rj.x - ri.x;
	rij.y = rj.y - ri.y;
	rij.z = rj.z - ri.z;

	/* compute acceleration */
	dst_sqr = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z + eps;
	cube = dst_sqr * dst_sqr * dst_sqr;
	inv_sqrt = rsqrtf(cube) * rj.w;

	/* acceleration a_i */
	ai.x = rij.x * inv_sqrt;
	ai.y = rij.y * inv_sqrt;
	ai.z = rij.z * inv_sqrt;

	return ai;
}

/* calculate accelerations */
__global__ void acc(float4 *r, float4 *a, float eps, float g)
{
	/* dynamically allocated shared memory */
	extern __shared__ float4 shared[];
	/* acceleration a_i */
	float3 ai = make_float3(0.f, 0.f, 0.f), tmp;
	/* position particle i */
	float3 ri;
	/* particle i */
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int k, l;

	/* get position of particle i */
	ri.x = r[i].x;
	ri.y = r[i].y;
	ri.z = r[i].z;	

	/* loop over tiles */
	for (k = 0; k < gridDim.x; ++k) {
		/* load position and mass into shared memory */
		shared[threadIdx.x] = r[__mul24(k, blockDim.x) + threadIdx.x];
		__syncthreads();

		/* loop over particles in a tile */
		#pragma unroll 32
		for (l = 0; l < blockDim.x; ++l) {
			tmp = interaction(ri, shared[l], eps);
			ai.x += tmp.x;
			ai.y += tmp.y;
			ai.z += tmp.z;
		}

		/* wait for other threads to finish calculation */
		__syncthreads();
	}

	/* save acceleration a_i in global memory */
	a[i] = make_float4(ai.x*g, ai.y*g, ai.z*g, 0.f);
}

__global__ void copy_to_vbo(float4 *r, float3 *pos)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);

	pos[i].x = r[i].x;
	pos[i].y = r[i].y;
	pos[i].z = r[i].z;
}

void init();
void cleanup();

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key) {
	/* ESC */
	case 27:
		exit(0);
		break;
	/* reset */
	case 'r':
		init();
		hipMemcpy(r, r_host, N*sizeof(float4), hipMemcpyHostToDevice);
		hipMemcpy(v, v_host, N*sizeof(float4), hipMemcpyHostToDevice);
		CHECK_ERROR;
		break;
	/* increase gravity */
	case '+':
		G += 2.;
		break;
	/* decrease gravity */
	case '-':
		G -= 2.;
		if (G < 0.)
			G = 0.1;
		break;
	/* increase dampening */
	case 'i':
		DAMPENING -= 0.05;
		if (DAMPENING < 0.)
			DAMPENING = 0.05;
		break;
	/* decrease dampening */
	case 'd':
		DAMPENING += 0.1;
		if (DAMPENING > 1.)
			DAMPENING = 1.;
		break;
	/* increase softening */
	case 'S':
		EPS += 0.1;
		break;
	/* decrease softening */
	case 's':
		EPS -= 0.1;
		if (EPS < 0.)
			EPS = 0.1;
		break;
	/* reset to default parameters */
	case 'p':
		G = 10.;
		EPS = 1.;
		DAMPENING = 0.7;
		break;
	default:
		break;
	}
}

void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN) {
		mouse_buttons |= 1<<button;
	} else if (state == GLUT_UP) {
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;

	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1) {
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	} else if (mouse_buttons & 4) {
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void display()
{
	float3 *pos;
	size_t size = N*sizeof(float3);

	/* run kernel */
	hipGraphicsMapResources(1, &vbo_res, 0);
	hipGraphicsResourceGetMappedPointer((void **)&pos, &size, vbo_res);
	acc<<<N/P, P>>>(r, a, EPS, G);
	leap_frog_1p_2<<<N/TPB, TPB>>>(a, v, r, DELTA_T, DAMPENING);
	copy_to_vbo<<<N/P, P>>>(r, pos);
	hipGraphicsUnmapResources(1, &vbo_res, 0);

	/* rotate view */
	glClear(GL_COLOR_BUFFER_BIT);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0., 0., translate_z);
	glRotatef(rotate_x, 1., 0., 0.);
	glRotatef(rotate_y, 0., 1., 0.);

	/* draw points */	
	glPointSize(1.);
	glColor4f(1., 1., 1., 1.);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_pos);
	glVertexPointer(3, GL_FLOAT, 0, 0);
	glEnableClientState(GL_VERTEX_ARRAY);
	glDrawArrays(GL_POINTS, 0, N);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();
	glutPostRedisplay();
}

int main(int argc, char *argv[])
{
	struct hipDeviceProp_t dev_prop;
	int device;
	int i;

	if (argc < 2) {
		printf("usage: nbody -N#bodies [-T#threads/block] -P#tiles\n");
		exit(1);
	}

	/* get command line parameters */
	for (i = 1; i < argc; ++i) {
		if (argv[i][0] == '-') {
			switch (argv[i][1]) {
			case 'N':
				N = atoi(argv[i]+2);
				break;
			case 'T':
				TPB = atoi(argv[i]+2);
				break;
			case 'P':
				P = atoi(argv[i]+2);
				break;
			default:
				break;
			}
		}
	}

	/*printf("N: %i, TPB: %i, TIMESTEPS: %i, P: %i\n", N, TPB, TIMESTEPS, P);*/

	if (N % TPB) {
		printf("#bodies must be a multiple of #threads/block!\n");
		exit(1);
	}

	if (N % P) {
		printf("#bodies must be a multiple of #p!\n");
		exit(1);
	}

	/* init OpenGL */
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("N-Body");
	glutDisplayFunc(display);
	/*glutReshapeFunc(reshape);*/
	glutKeyboardFunc(keyboard);
	/*glutSpecialFunc();*/
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glewInit();

	/* init GL */
	glClearColor(0., 0., 0., 1.);
	glDisable(GL_DEPTH_TEST);
	glViewport(0, 0, window_width, window_height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60., (GLfloat)window_width / (GLfloat)window_height,
	               1., 100.);

	/* get CUDA device properties */
	hipGetDevice(&device);
	hipGetDeviceProperties(&dev_prop, device);
	cudaGLSetGLDevice(device);
	CHECK_ERROR;

	/* create VBO */
	glGenBuffers(1, &vbo_pos);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_pos);
	glBufferData(GL_ARRAY_BUFFER, N*sizeof(float3), 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&vbo_res, vbo_pos,
                                     cudaGraphicsMapFlagsWriteDiscard);
	CHECK_ERROR;

	/* alloc host memory */
	r_host = (float4 *)malloc(N*sizeof(float4));
	v_host = (float4 *)malloc(N*sizeof(float4));
	/* alloc device memory */
	hipMalloc((void **)&a, N*sizeof(float4));
	hipMalloc((void **)&r, N*sizeof(float4));
	hipMalloc((void **)&v, N*sizeof(float4));
	CHECK_ERROR;

	/* generate initial configuration */
	T = gsl_rng_default;
	rng = gsl_rng_alloc(T);
	gsl_rng_set(rng, time(0));
	init();

	/* copy config to device memory */
	hipMemcpy(r, r_host, N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(v, v_host, N*sizeof(float4), hipMemcpyHostToDevice);
	CHECK_ERROR;

	atexit(cleanup);

	glutMainLoop();
	return 0;
}

/* generate initial configuration */
void init()
{
	int i;

	for (i = 0; i < N; ++i) {
		/* mass */
		r_host[i].w = gsl_rng_uniform(rng)>0.5 ? 1. : 10.;
		/*r_host[i].w = 5.;*/

		/* velocity */
		v_host[i].x = gsl_rng_uniform(rng) * 10.;
		v_host[i].y = gsl_rng_uniform(rng) * 5.;
		v_host[i].z = -3. * gsl_rng_uniform(rng);

		/* position */
		r_host[i].x = gsl_rng_uniform(rng) * 50.;
		r_host[i].y = gsl_rng_uniform(rng) * 50.;
		r_host[i].z = gsl_rng_uniform(rng) * 50.;
	}
}

void cleanup()
{
	/* free host memory */
	free(r_host);
	free(v_host);
	/* free device memory */
	hipFree(a);
	hipFree(r);
	hipFree(v);
	CHECK_ERROR;

	/* delete VBO */
	hipGraphicsUnregisterResource(vbo_res);
	glBindBuffer(1, vbo_pos);
	glDeleteBuffers(1, &vbo_pos);

	gsl_rng_free(rng);
}
