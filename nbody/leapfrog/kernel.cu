
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* check for CUDA error */
#define CHECK_ERROR check_cuda_error(__LINE__-1, __FILE__)

/* #bodies */
static int N;

/* #threads/block */
static int TPB = 128;

/* #timesteps */
static int TIMESTEPS = 1000;
/* softening factor (square), G, \Delta t */
static const float /*EPS = 0.1f, G = 2.f,*/ DELTA_T = 0.01f;

/* acceleration */
static float4 *a_host;
__device__ float4 *a;
/* x,y,z: position; w: mass */
static float4 *r_host;
__device__ float4 *r;
/* velocity */
static float4 *v_host;
__device__ float4 *v;

/* random number in [0,1] */
static inline float rnd()
{
	return (float)rand() / RAND_MAX;
}

/* check for CUDA error */
static void check_cuda_error(const int line, const char *file)
{
	hipError_t e;

	e = hipGetLastError();
	if (e != hipSuccess) {
		printf("CUDA error: %s, line %i, file '%s'\n",
		       hipGetErrorString(e), line, file);
		exit(1);
	}
}

/* leap frog integration kernel (2 particles/thread) */
__global__ void leap_frog_2p_1(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = (threadIdx.x + __mul24(blockIdx.x, blockDim.x)) << 1;

	v[i].x += a[i].x * delta_t;
	v[i].y += a[i].y * delta_t;
	v[i].z += a[i].z * delta_t;

	r[i].x += v[i].x * delta_t;
	r[i].y += v[i].y * delta_t;
	r[i].z += v[i].z * delta_t;

	v[i+1].x += a[i+1].x * delta_t;
	v[i+1].y += a[i+1].y * delta_t;
	v[i+1].z += a[i+1].z * delta_t;

	r[i+1].x += v[i+1].x * delta_t;
	r[i+1].y += v[i+1].y * delta_t;
	r[i+1].z += v[i+1].z * delta_t;
}

/* leap frog integration kernel (2 particles/thread) */
__global__ void leap_frog_2p_2(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = (threadIdx.x + __mul24(blockIdx.x, blockDim.x)) << 1;
	float3 v1, v2;

	v1.x = v[i].x;
	v1.y = v[i].y;
	v1.z = v[i].z;

	v2.x = v[i+1].x;
	v2.y = v[i+1].y;
	v2.z = v[i+1].z;

	v1.x += a[i].x * delta_t;
	v1.y += a[i].y * delta_t;
	v1.z += a[i].z * delta_t;

	r[i].x += v1.x * delta_t;
	r[i].y += v1.y * delta_t;
	r[i].z += v1.z * delta_t;

	v2.x += a[i+1].x * delta_t;
	v2.y += a[i+1].y * delta_t;
	v2.z += a[i+1].z * delta_t;

	r[i+1].x += v2.x * delta_t;
	r[i+1].y += v2.y * delta_t;
	r[i+1].z += v2.z * delta_t;

	v[i] = make_float4(v1.x, v1.y, v1.z, 0.f);
	v[i+1] = make_float4(v2.x, v2.y, v2.z, 0.f);
}

/* leap frog integration kernel (1 particle/thread) */
__global__ void leap_frog_1p_1(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);

	v[i].x += a[i].x * delta_t;
	v[i].y += a[i].y * delta_t;
	v[i].z += a[i].z * delta_t;

	r[i].x += v[i].x * delta_t;
	r[i].y += v[i].y * delta_t;
	r[i].z += v[i].z * delta_t;
}

/* leap frog integration kernel (1 particle/thread) */
__global__ void leap_frog_1p_2(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	float3 v_tmp;

	v_tmp.x = v[i].x;
	v_tmp.y = v[i].y;
	v_tmp.z = v[i].z;

	v_tmp.x += a[i].x * delta_t;
	v_tmp.y += a[i].y * delta_t;
	v_tmp.z += a[i].z * delta_t;

	r[i].x += v_tmp.x * delta_t;
	r[i].y += v_tmp.y * delta_t;
	r[i].z += v_tmp.z * delta_t;

	v[i] = make_float4(v_tmp.x, v_tmp.y, v_tmp.z, 0.f);
}

/* leap frog integration kernel (1 particle/thread) */
__global__ void leap_frog_1p_3(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	extern __shared__ float3 v_tmp[];

	v_tmp[threadIdx.x].x = v[i].x;
	v_tmp[threadIdx.x].y = v[i].y;
	v_tmp[threadIdx.x].z = v[i].z;

	v_tmp[threadIdx.x].x += a[i].x * delta_t;
	v_tmp[threadIdx.x].y += a[i].y * delta_t;
	v_tmp[threadIdx.x].z += a[i].z * delta_t;

	r[i].x += v_tmp[threadIdx.x].x * delta_t;
	r[i].y += v_tmp[threadIdx.x].y * delta_t;
	r[i].z += v_tmp[threadIdx.x].z * delta_t;

	v[i] = make_float4(v_tmp[threadIdx.x].x, v_tmp[threadIdx.x].y,
	                   v_tmp[threadIdx.x].z, 0.f);
}

void init();

int main(int argc, char *argv[])
{
	hipEvent_t start, stop;
	float time;
	int i, timestep;

	if (argc < 2) {
		printf("usage: leapfrog -N#bodies [-T#threads/block] [-S#timesteps]\n");
		exit(1);
	}

	/* get command line parameters */
	for (i = 1; i < argc; ++i) {
		if (argv[i][0] == '-') {
			switch (argv[i][1]) {
			case 'N':
				N = atoi(argv[i]+2);
				break;
			case 'T':
				TPB = atoi(argv[i]+2);
				break;
			case 'S':
				TIMESTEPS = atoi(argv[i]+2);
				break;
			default:
				break;
			}
		}
	}

	/*printf("N: %i, TPB: %i, TIMESTEPS: %i\n", N, TPB, TIMESTEPS);*/

	if (N % TPB) {
		printf("#bodies must be a multiple of #threads/block!\n");
		exit(1);
	}

	/* alloc host memory */
	a_host = (float4 *)malloc(N*sizeof(float4));
	r_host = (float4 *)malloc(N*sizeof(float4));
	v_host = (float4 *)malloc(N*sizeof(float4));
	/* alloc device memory */
	hipMalloc((void **)&a, N*sizeof(float4));
	hipMalloc((void **)&r, N*sizeof(float4));
	hipMalloc((void **)&v, N*sizeof(float4));
	CHECK_ERROR;

	/* generate initial configuration */
	srand(1);
	init();

	/* copy config to device memory */
	hipMemcpy(a, a_host, N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(r, r_host, N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(v, v_host, N*sizeof(float4), hipMemcpyHostToDevice);
	CHECK_ERROR;

	/* start counter */
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	/* integration steps */
	for (timestep = 0; timestep < TIMESTEPS; ++timestep) {
		/* update accelerations */

		/* leap frog */
		/*leap_frog_1p_1<<<N/TPB, TPB>>>(a, v, r, DELTA_T);*/
		leap_frog_1p_2<<<N/TPB, TPB>>>(a, v, r, DELTA_T);
		/*leap_frog_2p_1<<<N/TPB/2, TPB>>>(a, v, r, DELTA_T);*/
		/*leap_frog_2p_2<<<N/TPB/2, TPB>>>(a, v, r, DELTA_T);*/
		/*cudaMemcpy(r_host, r, N * sizeof(float4), cudaMemcpyDeviceToHost);
		printf("#1: x: %f, y: %f, z: %f\n", r_host[0].x, r_host[0].y, r_host[0].z);
		printf("#2: x: %f, y: %f, z: %f\n", r_host[1].x, r_host[1].y, r_host[1].z);*/
	}

	/* stop counter */
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	/* unit: milliseconds */
	hipEventElapsedTime(&time, start, stop);
	CHECK_ERROR;
	/*printf("elapsed time: %f ms\n", time);*/
	printf("%f\n", time);

	/* free host memory */
	free(a_host);
	free(r_host);
	free(v_host);
	/* free device memory */
	hipFree(a);
	hipFree(r);
	hipFree(v);
	CHECK_ERROR;

	return 0;
}

/* generate initial configuration */
void init()
{
	int i;

	for (i = 0; i < N; ++i) {
		/* mass */
		r_host[i].w = rnd()>0.5 ? 1.f : 10.f;

		/* velocity */
		v_host[i].x = 3.f;
		v_host[i].y = rnd() * 10.f;
		v_host[i].z = -5.f;

		/* position */
		r_host[i].x = rnd() * 50.f;
		r_host[i].y = rnd() * 50.f;
		r_host[i].z = rnd() * 50.f;
	}
}
