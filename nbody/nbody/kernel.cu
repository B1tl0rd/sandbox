
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* check for CUDA error */
#define CHECK_ERROR check_cuda_error(__LINE__-1, __FILE__)

/* #bodies */
static int N;

/* #threads/block (leapfrog) */
static int TPB = 128;

/* #tiles (acceleration kernel) */
static int P;

/* #timesteps */
static int TIMESTEPS = 1000;
/* softening factor (square), G, \Delta t */
static const float EPS = 0.1f, G = 2.f, DELTA_T = 0.01f;

/* acceleration */
__device__ float4 *a;
/* x,y,z: position; w: mass */
static float4 *r_host;
__device__ float4 *r;
/* velocity */
static float4 *v_host;
__device__ float4 *v;

/* random number in [0,1] */
static inline float rnd()
{
	return (float)rand() / RAND_MAX;
}

/* check for CUDA error */
static void check_cuda_error(const int line, const char *file)
{
	hipError_t e;

	e = hipGetLastError();
	if (e != hipSuccess) {
		printf("CUDA error: %s, line %i, file '%s'\n",
		       hipGetErrorString(e), line, file);
		exit(1);
	}
}

/* leap frog integration kernel (1 particle/thread) */
__global__ void leap_frog_1p_2(float4 *a, float4 *v, float4 *r, float delta_t)
{
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	float3 v_tmp;

	v_tmp.x = v[i].x;
	v_tmp.y = v[i].y;
	v_tmp.z = v[i].z;

	v_tmp.x += a[i].x * delta_t;
	v_tmp.y += a[i].y * delta_t;
	v_tmp.z += a[i].z * delta_t;

	r[i].x += v_tmp.x * delta_t;
	r[i].y += v_tmp.y * delta_t;
	r[i].z += v_tmp.z * delta_t;

	v[i] = make_float4(v_tmp.x, v_tmp.y, v_tmp.z, 0.f);
}

/* body-body interaction, returns a_i */
__device__ float3 interaction(float3 ri, float4 rj, float eps)
{
	float3 rij, ai;
	float dst_sqr, cube, inv_sqrt;

	/* distance vector */
	rij.x = rj.x - ri.x;
	rij.y = rj.y - ri.y;
	rij.z = rj.z - ri.z;

	/* compute acceleration */
	dst_sqr = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z + eps;
	cube = dst_sqr * dst_sqr * dst_sqr;
	inv_sqrt = rsqrtf(cube) * rj.w;

	/* acceleration a_i */
	ai.x = rij.x * inv_sqrt;
	ai.y = rij.y * inv_sqrt;
	ai.z = rij.z * inv_sqrt;

	return ai;
}

/* calculate accelerations */
__global__ void acc(float4 *r, float4 *a, float eps, float g)
{
	/* dynamically allocated shared memory */
	extern __shared__ float4 shared[];
	/* acceleration a_i */
	float3 ai = make_float3(0.f, 0.f, 0.f), tmp;
	/* position particle i */
	float3 ri;
	/* particle i */
	int i = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int k, l;

	/* get position of particle i */
	ri.x = r[i].x;
	ri.y = r[i].y;
	ri.z = r[i].z;	

	/* loop over tiles */
	for (k = 0; k < gridDim.x; ++k) {
		/* load position and mass into shared memory */
		shared[threadIdx.x] = r[__mul24(k, blockDim.x) + threadIdx.x];
		__syncthreads();

		/* loop over particles in a tile */
		#pragma unroll 32
		for (l = 0; l < blockDim.x; ++l) {
			tmp = interaction(ri, shared[l], eps);
			ai.x += tmp.x;
			ai.y += tmp.y;
			ai.z += tmp.z;
		}

		/* wait for other threads to finish calculation */
		__syncthreads();
	}

	/* save acceleration a_i in global memory */
	a[i] = make_float4(ai.x*g, ai.y*g, ai.z*g, 0.f);
}

void init();

int main(int argc, char *argv[])
{
	hipEvent_t start, stop;
	float time;
	int i, timestep;

	if (argc < 2) {
		printf("usage: nbody -N#bodies [-T#threads/block] [-S#timesteps] -P#tiles\n");
		exit(1);
	}

	/* get command line parameters */
	for (i = 1; i < argc; ++i) {
		if (argv[i][0] == '-') {
			switch (argv[i][1]) {
			case 'N':
				N = atoi(argv[i]+2);
				break;
			case 'T':
				TPB = atoi(argv[i]+2);
				break;
			case 'S':
				TIMESTEPS = atoi(argv[i]+2);
				break;
			case 'P':
				P = atoi(argv[i]+2);
				break;
			default:
				break;
			}
		}
	}

	/*printf("N: %i, TPB: %i, TIMESTEPS: %i, P: %i\n", N, TPB, TIMESTEPS, P);*/

	if (N % TPB) {
		printf("#bodies must be a multiple of #threads/block!\n");
		exit(1);
	}

	if (N % P) {
		printf("#bodies must be a multiple of #p!\n");
		exit(1);
	}

	/* alloc host memory */
	r_host = (float4 *)malloc(N*sizeof(float4));
	v_host = (float4 *)malloc(N*sizeof(float4));
	/* alloc device memory */
	hipMalloc((void **)&a, N*sizeof(float4));
	hipMalloc((void **)&r, N*sizeof(float4));
	hipMalloc((void **)&v, N*sizeof(float4));
	CHECK_ERROR;

	/* generate initial configuration */
	srand(1);
	init();

	/* copy config to device memory */
	hipMemcpy(r, r_host, N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(v, v_host, N*sizeof(float4), hipMemcpyHostToDevice);
	CHECK_ERROR;

	/* start counter */
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	/* integration steps */
	for (timestep = 0; timestep < TIMESTEPS; ++timestep) {
		/* update accelerations */
		acc<<<N/P, P>>>(r, a, EPS, G);

		/* leap frog */
		leap_frog_1p_2<<<N/TPB, TPB>>>(a, v, r, DELTA_T);
		/*cudaMemcpy(r_host, r, N * sizeof(float3), cudaMemcpyDeviceToHost);
		printf("#1: x: %f, y: %f, z: %f\n", r_host[0].x, r_host[0].y, r_host[0].z);
		printf("#2: x: %f, y: %f, z: %f\n", r_host[1].x, r_host[1].y, r_host[1].z);*/
	}

	/* stop counter */
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	/* unit: milliseconds */
	hipEventElapsedTime(&time, start, stop);
	CHECK_ERROR;
	/*printf("elapsed time: %f\n", time);
	printf("#interactions/s: %f\n", ((float)TIMESTEPS*N*N) / time * 1000);*/
	printf("%f\n", ((float)TIMESTEPS*N*N) / time * 1000);

	/* free host memory */
	free(r_host);
	free(v_host);
	/* free device memory */
	hipFree(a);
	hipFree(r);
	hipFree(v);
	CHECK_ERROR;

	return 0;
}

/* generate initial configuration */
void init()
{
	int i;

	for (i = 0; i < N; ++i) {
		/* mass */
		r_host[i].w = rnd()>0.5 ? 1.f : 10.f;

		/* velocity */
		v_host[i].x = 3.f;
		v_host[i].y = rnd() * 10.f;
		v_host[i].z = -5.f;

		/* position */
		r_host[i].x = rnd() * 50.f;
		r_host[i].y = rnd() * 50.f;
		r_host[i].z = rnd() * 50.f;
	}
}
